#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <stdio.h>

#include "kernels.cuh"
#include ""

typedef unsigned char   uint8;
typedef unsigned int    uint32;
typedef int             int32;

extern "C"
    __device__ unsigned char clip_value(unsigned char x, unsigned char min_val, unsigned char  max_val) {
    if (x > max_val) {
        return max_val;
    }
    else if (x < min_val) {
        return min_val;
    }
    else {
        return x;
    }
}

extern "C"
    __global__ void kernel_rgb2yuv(signed short* src_img, unsigned char* Y, unsigned char* u, unsigned char* v,
        int src_width, int src_height, size_t yPitch, int rIntercept, int rSlope, short minWindowValue, short windowWidth)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= src_width)
        return; //x = width - 1;

    if (y >= src_height)
        return; // y = height - 1;

    // 1st
    // final_value = original_value * rescale_slope + rescale_intercept
    float _B = (float)(src_img[y * src_width + x] * rSlope + rIntercept);

    // 2nd
    //minWindowValue = windowLevel - (windowWidth / 2) 
    //jpegValue = 255 * (dicomValue - minWindowValue) / windowWidth

    float B= 255.0 * (_B - minWindowValue) / windowWidth;
    float G = B;
    float R = B;

    Y[y * yPitch + x] = clip_value((unsigned char)(0.299 * R + 0.587 * G + 0.114 * B), 0, 255);
    u[y * src_width + x] = clip_value((unsigned char)(-0.147 * R - 0.289 * G + 0.436 * B + 128), 0, 255);
    v[y * src_width + x] = clip_value((unsigned char)(0.615 * R - 0.515 * G - 0.100 * B + 128), 0, 255);
}

extern "C"
    __global__ void kernel_resize_UV(unsigned char* src_img, unsigned char *dst_img,
        int src_width, int src_height, int dst_width, int dst_height, int nPitch)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst_width)
        return; //x = width - 1;

    if (y >= dst_height)
        return; // y = height - 1;

    float fx = (x + 0.5)*src_width / (float)dst_width - 0.5;
    float fy = (y + 0.5)*src_height / (float)dst_height - 0.5;
    int ax = floor(fx);
    int ay = floor(fy);
    if (ax < 0)
    {
        ax = 0;
    }
    else if (ax > src_width - 2)
    {
        ax = src_width - 2;
    }

    if (ay < 0) 
    {
        ay = 0;
    }
    else if (ay > src_height - 2)
    {
        ay = src_height - 2;
    }

    int A = ax + ay*src_width;
    int B = ax + ay*src_width + 1;
    int C = ax + ay*src_width + src_width;
    int D = ax + ay*src_width + src_width + 1;

    float w1, w2, w3, w4;
    w1 = fx - ax;
    w2 = 1 - w1;
    w3 = fy - ay;
    w4 = 1 - w3;

    unsigned char val = src_img[A] * w2*w4 + src_img[B] * w1*w4 + src_img[C] * w2*w3 + src_img[D] * w1*w3;

    dst_img[y * nPitch + x] = clip_value(val, 0, 255);
}
